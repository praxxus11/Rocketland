// SHOULD BE COMPIL ED WITH NVCC SO NO INCLUDES
#include "matmul.h"
#include <stdio.h>
#include <iostream>

#define checkCudaErrors(call)                                      \
    do {                                                           \
        hipError_t err = call;                                    \
        if (err != hipSuccess) {                                  \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__,\
                hipGetErrorString(err));                          \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    } while (0)


void matmul(
    const float* weights,  const float* biases, 
    float* inputs, float* outputs, 
    int num_layers, const int* l_sizes,
    int batches) {

    int weight_bytes = 0;
    int bias_bytes = 0;
    int input_bytes = l_sizes[0];
    int output_bytes = l_sizes[num_layers-1];

    for (int i=0; i<num_layers-1; i++) {
        weight_bytes += l_sizes[i] * l_sizes[i+1];
        bias_bytes += l_sizes[i+1];
    }

    weight_bytes *= batches * sizeof(float);
    bias_bytes *= batches * sizeof(float);
    input_bytes *= batches * sizeof(float);
    output_bytes *= batches* sizeof(float);

    float* d_weights;
    float* d_biases;
    float* d_inputs;
    float* d_outputs;

    checkCudaErrors(hipMalloc(&d_weights, weight_bytes));
    checkCudaErrors(hipMalloc(&d_biases, bias_bytes));
    checkCudaErrors(hipMalloc(&d_inputs, input_bytes));

    checkCudaErrors(hipMemcpy(d_weights, weights, weight_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_biases, biases, bias_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inputs, inputs, input_bytes, hipMemcpyHostToDevice));

    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed.");
        return;
    }

    int weights_so_far = 0;
    for (int layer = 0; layer < num_layers - 1; layer++) {
        int curr_output_bytes = l_sizes[layer + 1] * batches * sizeof(float);
        checkCudaErrors(hipMalloc(&d_outputs, curr_output_bytes));
        
        
        const float m = 1.f;
        const float n = l_sizes[layer + 1];
        const float k = l_sizes[layer];

        const float alpha = 1.f;
        const float lda = m;
        const long long int strideA = l_sizes[layer];

        const float ldb = k;
        const long long int strideB = weight_bytes / (sizeof(float) * batches);
        const long long int startB = weights_so_far;

        const float beta = 0.f;

        const float ldc = m;
        const long long int strideC = l_sizes[layer + 1];

        status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            m, n, k,
                            &alpha, d_inputs, lda, strideA, 
                            d_weights + startB, ldb, strideB,
                            &beta, d_outputs, ldc, strideC,
                            batches);
        hipDeviceSynchronize();
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Failed.");
            return;
        }

        if (layer < num_layers - 2) {
            checkCudaErrors(hipFree(d_inputs));
            checkCudaErrors(hipMalloc(&d_inputs, curr_output_bytes));
            checkCudaErrors(hipMemcpy(d_inputs, d_outputs, curr_output_bytes, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipFree(d_outputs));
        }
        weights_so_far += l_sizes[layer+1] * l_sizes[layer];
    }
    checkCudaErrors(hipMemcpy(outputs, d_outputs, output_bytes, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_weights));
    checkCudaErrors(hipFree(d_biases));
    checkCudaErrors(hipFree(d_inputs));
    checkCudaErrors(hipFree(d_outputs));
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed.");
        return;
    }
}