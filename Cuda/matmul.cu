#include "hip/hip_runtime.h"
// SHOULD BE COMPIL ED WITH NVCC SO NO INCLUDES
#include "matmul.h"
#include <stdio.h>
// #include <hipblas.h>
#include <iostream>

#define checkCudaErrors(call)                                      \
    do {                                                           \
        hipError_t err = call;                                    \
        if (err != hipSuccess) {                                  \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__,\
                hipGetErrorString(err));                          \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    } while (0)

__global__ void elemwise_tanh(float* activations, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        activations[i] = tanh(activations[i]);
    }
}

__global__ void biases_add(float* activations, float* biases, int batches, int activations_per_batch, int bias_stride, int bias_start) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (; i < batches; i += stride) { // i represents which batch you're on
        for (int j = 0; j < activations_per_batch; j++) {
            activations[i * activations_per_batch + j] += biases[i * bias_stride + j + bias_start];
        }
    }
}

void matmul(
    const float* weights,  const float* biases, 
    float* inputs, float* outputs, 
    int num_layers, const int* l_sizes,
    int batches) {

    int weight_bytes = 0;
    int bias_bytes = 0;
    int input_bytes = l_sizes[0];
    int output_bytes = l_sizes[num_layers-1];

    for (int i=0; i<num_layers-1; i++) {
        weight_bytes += l_sizes[i] * l_sizes[i+1];
        bias_bytes += l_sizes[i+1];
    }

    weight_bytes *= batches * sizeof(float);
    bias_bytes *= batches * sizeof(float);
    input_bytes *= batches * sizeof(float);
    output_bytes *= batches* sizeof(float);

    float* d_weights;
    float* d_biases;
    float* d_inputs;
    float* d_outputs;

    checkCudaErrors(hipMalloc(&d_weights, weight_bytes));
    checkCudaErrors(hipMalloc(&d_biases, bias_bytes));
    checkCudaErrors(hipMalloc(&d_inputs, input_bytes));

    checkCudaErrors(hipMemcpy(d_weights, weights, weight_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_biases, biases, bias_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inputs, inputs, input_bytes, hipMemcpyHostToDevice));

    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed.");
        return;
    }

    int weights_so_far = 0;
    int biases_so_far = 0;
    for (int layer = 0; layer < num_layers - 1; layer++) {
        int curr_output_bytes = l_sizes[layer + 1] * batches * sizeof(float);
        checkCudaErrors(hipMalloc(&d_outputs, curr_output_bytes));
        
        
        const float m = 1.f;
        const float n = l_sizes[layer+1];
        const float k = l_sizes[layer];

        const float alpha = 1.f;
        const long long int strideA = l_sizes[layer];

        const long long int strideB = weight_bytes / (sizeof(float) * batches);
        const long long int startB = weights_so_far;

        const float beta = 0.f;

        const long long int strideC = l_sizes[layer + 1];

        // first do all matrix multiplications
        status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
                                            m, n, k,
                                            &alpha, d_inputs, k, strideA, 
                                            d_weights + startB, n, strideB,
                                            &beta, d_outputs, m, strideC,
                                            batches);
        hipDeviceSynchronize();

        // add biases onto it
        const int threads_biasadd = 64;
        const int blocks_biasadd = (batches + threads_biasadd - 1) / threads_biasadd;
        biases_add<<<blocks_biasadd, threads_biasadd>>>(d_outputs, 
                                                        d_biases, 
                                                        batches, 
                                                        l_sizes[layer + 1], 
                                                        bias_bytes / (batches * sizeof(float)),
                                                        biases_so_far);
        hipDeviceSynchronize();

        // appply activation func
        const int threads_tanh = 256;
        const int blocks_tanh = ((l_sizes[layer + 1] * batches) + threads_tanh - 1) / threads_tanh;
        elemwise_tanh<<<blocks_tanh, threads_tanh>>>(d_outputs, l_sizes[layer + 1] * batches);
        hipDeviceSynchronize();

        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Failed.");
            return;
        }

        if (layer < num_layers - 2) {
            checkCudaErrors(hipFree(d_inputs));
            checkCudaErrors(hipMalloc(&d_inputs, curr_output_bytes));
            checkCudaErrors(hipMemcpy(d_inputs, d_outputs, curr_output_bytes, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipFree(d_outputs));
        }
        weights_so_far += l_sizes[layer+1] * l_sizes[layer];
        biases_so_far += l_sizes[layer+1];
    }
    checkCudaErrors(hipMemcpy(outputs, d_outputs, output_bytes, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_weights));
    checkCudaErrors(hipFree(d_biases));
    checkCudaErrors(hipFree(d_inputs));
    checkCudaErrors(hipFree(d_outputs));
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed.");
        return;
    }
}