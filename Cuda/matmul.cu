#include "hip/hip_runtime.h"
// SHOULD BE COMPIL ED WITH NVCC SO NO INCLUDES
#include "matmul.h"
#include <stdio.h>
// include <hipblas.h>
#include <iostream>

#define checkCudaErrors(call)                                      \
    do {                                                           \
        hipError_t err = call;                                    \
        if (err != hipSuccess) {                                  \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__,\
                hipGetErrorString(err));                          \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    } while (0)

__global__ void elemwise_tanh(float* activations, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        activations[i] = tanh(activations[i]);
    }
}

void matmul(
    const float* weights,  const float* biases, 
    float* inputs, float* outputs, 
    int num_layers, const int* l_sizes,
    int batches) {

    int weight_bytes = 0;
    int bias_bytes = 0;
    int input_bytes = l_sizes[0];
    int output_bytes = l_sizes[num_layers-1];

    for (int i=0; i<num_layers-1; i++) {
        weight_bytes += l_sizes[i] * l_sizes[i+1];
        bias_bytes += l_sizes[i+1];
    }

    weight_bytes *= batches * sizeof(float);
    bias_bytes *= batches * sizeof(float);
    input_bytes *= batches * sizeof(float);
    output_bytes *= batches* sizeof(float);

    float* d_weights;
    float* d_biases;
    float* d_inputs;
    float* d_outputs;

    checkCudaErrors(hipMalloc(&d_weights, weight_bytes));
    checkCudaErrors(hipMalloc(&d_biases, bias_bytes));
    checkCudaErrors(hipMalloc(&d_inputs, input_bytes));

    checkCudaErrors(hipMemcpy(d_weights, weights, weight_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_biases, biases, bias_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inputs, inputs, input_bytes, hipMemcpyHostToDevice));

    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed.");
        return;
    }

    int weights_so_far = 0;
    for (int layer = 0; layer < num_layers - 1; layer++) {
        int curr_output_bytes = l_sizes[layer + 1] * batches * sizeof(float);
        checkCudaErrors(hipMalloc(&d_outputs, curr_output_bytes));
        
        
        const float m = 1.f;
        const float n = l_sizes[layer+1];
        const float k = l_sizes[layer];

        const float alpha = 1.f;
        const long long int strideA = l_sizes[layer];

        const long long int strideB = weight_bytes / (sizeof(float) * batches);
        const long long int startB = weights_so_far;

        const float beta = 0.f;

        const long long int strideC = l_sizes[layer + 1];

        // first do all matrix multiplications
        status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
                            m, n, k,
                            &alpha, d_inputs, k, strideA, 
                            d_weights + startB, n, strideB,
                            &beta, d_outputs, m, strideC,
                            batches);
        hipDeviceSynchronize();


        // appply activation func
        const int activations_len = l_sizes[layer + 1] * batches;
        const int threads_tanh = 256;
        const int blocks_tanh = (activations_len + threads_tanh - 1) / threads_tanh;
        elemwise_tanh<<<blocks_tanh, threads_tanh>>>(d_outputs, l_sizes[layer + 1] * batches);
        hipDeviceSynchronize();

        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Failed.");
            return;
        }

        if (layer < num_layers - 2) {
            checkCudaErrors(hipFree(d_inputs));
            checkCudaErrors(hipMalloc(&d_inputs, curr_output_bytes));
            checkCudaErrors(hipMemcpy(d_inputs, d_outputs, curr_output_bytes, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipFree(d_outputs));
        }
        weights_so_far += l_sizes[layer+1] * l_sizes[layer];
    }
    checkCudaErrors(hipMemcpy(outputs, d_outputs, output_bytes, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_weights));
    checkCudaErrors(hipFree(d_biases));
    checkCudaErrors(hipFree(d_inputs));
    checkCudaErrors(hipFree(d_outputs));
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed.");
        return;
    }
}