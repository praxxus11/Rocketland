// SHOULD BE COMPIL ED WITH NVCC SO NO INCLUDES
#include "matmul.h"

void matmul(
    float* weights, float* biases, 
    float* inputs, float* outputs, 
    int num_layers, int* l_sizes,
    int batches) {

    int weight_bytes = 0;
    int bias_bytes = 0;
    int input_bytes = l_sizes[0];
    int output_bytes = l_sizes[num_layers-1];

    for (int i=0; i<num_layers-1; i++) {
        weight_bytes += l_sizes[i] * l_sizes[i+1];
        bias_bytes += l_sizes[i+1];
    }
    weight_bytes *= batches * sizeof(float);
    bias_bytes *= batches * sizeof(float);
    input_bytes *= batches * sizeof(float);
    output_bytes *= batches* sizeof(float);

    float* d_weights;
    float* d_biases;
    float* d_inputs;

    checkCudaErrors(hipMalloc(&d_weights, weight_bytes));
    checkCudaErrors(hipMalloc(&d_biases, bias_bytes));
    checkCudaErrors(hipMalloc(&d_inputs, input_bytes));

    checkCudaErrors(hipMemcpy(d_weights, weights, weight_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_biases, biases, bias_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inputs, inputs, input_bytes, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    checkCudaErrors(hipblasCreate(&handle));


    int weights_so_far = 0;
    for (int layer = 0; layer < num_layers; layer++) {
        float* intermediate;
        int intermediate_bytes = l_sizes[layer + 1] * batches * sizeof(float);
        checkCudaErrors(hipMalloc(&intermediate, intermediate_bytes));
        
        const float m = 1.f;
        const float n = l_sizes[layer + 1];
        const float k = l_sizes[layer];

        const float alpha = 1.f;
        const float lda = m;
        const long long int strideA = l_sizes[layer];

        const float ldb = k;
        const long long int strideB = weight_bytes / (sizeof(float) * batches);
        const long long int startB = weights_so_far;

        const float beta = 0.f;

        const float ldc = m;
        const long long int strideC = l_sizes[layer + 1];

        hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            m, n, k,
                            alpha, inputs, lda, strideA, 
                            weights + startB, ldb, strideB,
                            beta, intermediate, ldc, strideC,
                            batches);
        weights_so_far += l_sizes[layer+1] * l_sizes[layer];
    }


    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_inputs);
    hipblasDestroy(handle);
}